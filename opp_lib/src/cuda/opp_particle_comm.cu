#include "hip/hip_runtime.h"
/* 
BSD 3-Clause License

Copyright (c) 2022, OP-DSL

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// #pragma once

#include "opp_cuda.h"

#define MPI_COUNT_EXCHANGE 0
#define MPI_TAG_PART_EX 1

//*******************************************************************************
void opp_part_pack_device(opp_set set);
void opp_part_unpack_device(opp_set set);
void particle_sort_device(opp_set set, bool hole_filling);
void particle_hole_fill_device(opp_set set);
std::vector<char> OPP_need_remove_flags;
char *OPP_need_remove_flags_d = nullptr;
int OPP_need_remove_flags_size = 0;

thrust::device_vector<int> OPP_thrust_move_particle_indices_d;
thrust::device_vector<int> OPP_thrust_move_cell_indices_d;
int *OPP_move_particle_indices_d = nullptr;
int *OPP_move_cell_indices_d = nullptr;
int *OPP_move_count_d = nullptr;
int OPP_move_count_h = 0;

int *OPP_remove_particle_indices_d = nullptr;
thrust::device_vector<int> OPP_thrust_remove_particle_indices_d;
int *OPP_remove_count_d = nullptr;
int OPP_remove_count_h = 0;

// int OPP_move_indices_capacity = 0;

struct CopyMaxCellIndexFunctor {
    int* B;
    CopyMaxCellIndexFunctor(int* B) : B(B) {}

    __host__ __device__
    void operator()(int index) const {
        B[index] = MAX_CELL_INDEX;
    }
};

//*******************************************************************************
void opp_init_particle_move(opp_set set, int nargs, opp_arg *args)
{ 

    opp_init_particle_move_core(set);

    cutilSafeCall(hipMemcpy(set->particle_remove_count_d, &(set->particle_remove_count), sizeof(int), 
                    hipMemcpyHostToDevice));

    const int half_set_size_alloc_mul = (int)(set->size * OPP_part_alloc_mult / 2);
    if (half_set_size_alloc_mul > (int)OPP_thrust_move_particle_indices_d.size())
    {     
        OPP_thrust_move_particle_indices_d.resize(half_set_size_alloc_mul);
        OPP_thrust_move_cell_indices_d.resize(half_set_size_alloc_mul);

        OPP_move_particle_indices_d = (int*)thrust::raw_pointer_cast(OPP_thrust_move_particle_indices_d.data());
        OPP_move_cell_indices_d = (int*)thrust::raw_pointer_cast(OPP_thrust_move_cell_indices_d.data());

        if (OPP_move_count_d == nullptr) {
            cutilSafeCall(hipMalloc(&OPP_move_count_d, sizeof(int)));
        }

        OPP_thrust_remove_particle_indices_d.resize(half_set_size_alloc_mul);
        OPP_remove_particle_indices_d = (int*)thrust::raw_pointer_cast(OPP_thrust_remove_particle_indices_d.data());
        if (OPP_remove_count_d == nullptr) {
            cutilSafeCall(hipMalloc(&OPP_remove_count_d, sizeof(int)));
        }
    }

    OPP_move_count_h = 0;
    cutilSafeCall(hipMemcpy(OPP_move_count_d, &OPP_move_count_h, sizeof(int), hipMemcpyHostToDevice));

    if (OPP_comm_iteration == 0)
    {
        OPP_iter_start = 0;
        OPP_iter_end   = set->size;
        OPP_part_comm_count_per_iter = 0;    
    }
    else
    {
        // need to change the arg data since particle communication could change the pointer in realloc dat->data
        for (int i = 0; i < nargs; i++)
        {
            if (args[i].argtype == OPP_ARG_DAT && args[i].dat->set->is_particle)
            {
                args[i].data = args[i].dat->data;
                args[i].data_d = args[i].dat->data_d;
                // if (OPP_DBG) opp_printf("SSSS", "dat %s %p", args[i].dat->name, args[i].dat->data_d);
            }
        }
    }

    OPP_mesh_relation_data = ((int *)set->mesh_relation_dat->data); 
    OPP_mesh_relation_data_d = ((int *)set->mesh_relation_dat->data_d); 
}

//*******************************************************************************
// 1. keep track of whether atleast one need mpi comm [WRONG] I might not require to send to others, but some one might try to send to me 
// 2. if yes, download dats from device and use opp_part_exchange (in this particle will get copied so we can override the space)
// 3. do the below, opp_particle_sort or particle_sort_device
// 4. check whether all mpi ranks are done and if yes, return false
// 5. if no, wait for all to complete and copy only the received particles to the device buffer 
//        (may need to write another path and to expand device array sizes)
// 6. set new start and ends and return true
// bool opp_finalize_particle_move(opp_set set)
// { 
//     opp_profiler->start("Mv_Finalize");

//     cutilSafeCall(hipDeviceSynchronize());

//     OPP_move_count_h = 0;
//     cutilSafeCall(hipMemcpy(&OPP_move_count_h, OPP_move_count_d, sizeof(int), 
//         hipMemcpyDeviceToHost));

//     cutilSafeCall(hipMemcpy(&(set->particle_remove_count), set->particle_remove_count_d, 
//                     sizeof(int), hipMemcpyDeviceToHost));

//     if (OPP_DBG) //  || OPP_comm_iteration != 0
//         opp_printf("opp_finalize_particle_move", "set [%s][%d] remove_count [%d] move count [%d]", 
//             set->name, set->size, set->particle_remove_count, OPP_move_count_h);

// #ifdef USE_MPI
//     // At this stage, particles of device is clean

//     // download only the required particles to send and pack them in rank based mpi buffers
//     // opp_profiler->start("Mv_F_pack");
//     if (OPP_comm_iteration == 0) opp_profiler->start("Mv_F_pack0");
//     else if (OPP_comm_iteration == 1) opp_profiler->start("Mv_F_pack1");
//     else opp_profiler->start("Mv_F_pack");
//     opp_part_pack_device(set);
//     if (OPP_comm_iteration == 0) opp_profiler->end("Mv_F_pack0");
//     else if (OPP_comm_iteration == 1) opp_profiler->end("Mv_F_pack1");
//     else opp_profiler->end("Mv_F_pack");
//     // opp_profiler->end("Mv_F_pack");

//     // send the counts and send the particles  
//     if (OPP_comm_iteration == 0) opp_profiler->start("Mv_F_ex0");
//     else if (OPP_comm_iteration == 1) opp_profiler->start("Mv_F_ex1");
//     else opp_profiler->start("Mv_F_ex");
//     // opp_profiler->start("Mv_F_ex");     
//     opp_part_exchange(set); 
//     // opp_profiler->end("Mv_F_ex");
//     if (OPP_comm_iteration == 0) opp_profiler->end("Mv_F_ex0");
//     else if (OPP_comm_iteration == 1) opp_profiler->end("Mv_F_ex1");
//     else opp_profiler->end("Mv_F_ex");
// #endif

//     if (OPP_comm_iteration == 0) opp_profiler->start("Mv_F_fill0");
//     else if (OPP_comm_iteration == 1) opp_profiler->start("Mv_F_fill1");
//     else opp_profiler->start("Mv_F_fill");
//     // opp_profiler->start("Mv_F_fill");
//     if (set->particle_remove_count > 0)
//     {
//         set->size -= set->particle_remove_count;

//         if (OPP_auto_sort == 1)
//         {
//             if (OPP_DBG) 
//                 opp_printf("opp_finalize_particle_move", "auto sorting particle set [%s]", 
//                 set->name);
//             opp_particle_sort(set);
//         }
//         else
//         {
//             // if (opp_params->get<OPP_STRING>("fill") == "r")
//                 particle_sort_device(set, true); // Does only hole filling
//             // else
//             //     particle_hole_fill_cuda(set, true);
//         }
//     }
//     // opp_profiler->end("Mv_F_fill");
//     if (OPP_comm_iteration == 0) opp_profiler->end("Mv_F_fill0");
//     else if (OPP_comm_iteration == 1) opp_profiler->end("Mv_F_fill1");
//     else opp_profiler->end("Mv_F_fill");

// #ifdef USE_MPI
//     opp_profiler->start("Mv_F_check");
//     if (opp_part_check_all_done(set))
//     {
//         if (OPP_max_comm_iteration < OPP_comm_iteration)
//             OPP_max_comm_iteration = OPP_comm_iteration;

//         OPP_comm_iteration = 0; // reset for the next par loop
        
//         cutilSafeCall(hipDeviceSynchronize());
//         opp_profiler->end("Mv_Finalize");
//         opp_profiler->end("Mv_F_check");
//         return false; // all mpi ranks do not have anything to communicate to any rank
//     }
//     opp_profiler->end("Mv_F_check");

//     opp_profiler->start("Mv_F_wait");
//     opp_part_wait_all(set); // wait till all the particles are communicated
//     opp_profiler->end("Mv_F_wait");

//     if (OPP_DBG)
//         opp_printf("opp_finalize_particle_move", "set [%s] size prior unpack %d", set->name, set->size);
    
//     cutilSafeCall(hipDeviceSynchronize());

//     // increase the particle count if required and unpack the communicated particle buffer 
//     // in to separate particle dats
//     opp_profiler->start("Mv_F_unpack");
//     opp_part_unpack_device(set);    
//     opp_profiler->end("Mv_F_unpack");

//     OPP_iter_start = set->size - set->diff;
//     OPP_iter_end   = set->size;  

//     OPP_comm_iteration++;  

//     opp_profiler->end("Mv_Finalize");

//     return true;
// #else
//     return false;
// #endif
// }

thrust::device_vector<int> send_part_cell_idx_dv;
thrust::device_vector<int> temp_int_dv;
thrust::device_vector<double> temp_real_dv;

// Cannot use multiple packs before sending them, if opp_part_pack() is called multiple times with PACK_SOA, 
// the communication data may get currupted
//*******************************************************************************
void opp_part_pack_device(opp_set set)
{
    if (OPP_DBG) opp_printf("opp_part_pack_device", "start");

#ifdef USE_MPI
    opp_profiler->start("Mv_Pack");

    // auto findIfPredicate = [](char value) { return value == 1; };
    // auto findIfBegin = std::find_if(OPP_need_remove_flags.begin(), OPP_need_remove_flags.end(), findIfPredicate);
    // auto findIfEnd = OPP_need_remove_flags.end();
    // while (findIfBegin != findIfEnd) 
    // {
    //     send_indices_hv.push_back(std::distance(OPP_need_remove_flags.begin(), findIfBegin));
    //     findIfBegin = std::find_if(std::next(findIfBegin), findIfEnd, findIfPredicate);
    // }

    if (OPP_move_count_h <= 0) 
    {
        opp_profiler->end("Mv_Pack");
        return;
    }

    // Uncomment below if comparing cuda_mpi with sycl_mpi or hip_mpi
    if (debugger) {
        thrust::sort_by_key(cellIdx_dv.begin(), cellIdx_dv.end(), i_dv.begin());
            thrust::sort_by_key(
                OPP_thrust_move_particle_indices_d.begin(), OPP_thrust_move_particle_indices_d.begin() + OPP_move_count_h, 
                OPP_thrust_move_cell_indices_d.begin());
    }

    // Since cuda kernel threads are not synced, there could be a random order
    // thrust::sort(OPP_thrust_move_particle_indices_d.begin(), 
    //     OPP_thrust_move_particle_indices_d.begin() + OPP_move_count_h);

    // thrust::copy(OPP_thrust_move_particle_indices_d.begin(), 
    //     OPP_thrust_move_particle_indices_d.begin() + OPP_move_count_h, OPP_thrust_move_indices_h.begin());
    
    // std::string loggg = std::to_string(OPP_move_count_h) + " | ";
    // for (int k = 0; k < OPP_move_count_h; k++)
    //     loggg += std::to_string(OPP_thrust_move_indices_h[k]) + std::string(" ");
    // opp_printf("opp_part_pack_device", "Part indices : %s", loggg.c_str());

    // copy the cell indices of the particles to be sent
    // send_part_cell_idx_dv.reserve(OPP_move_count_h);
    // send_part_cell_idx_dv.resize(OPP_move_count_h);
    // copy_according_to_index(set->mesh_relation_dat->thrust_int, &send_part_cell_idx_dv, 
    //     OPP_thrust_move_particle_indices_d, -1, -1, OPP_move_count_h, 1);

    // opp_profiler->start("Mv_Pack1");
    thrust::host_vector<int> send_part_cell_idx_hv(OPP_move_count_h);
    thrust::copy(OPP_thrust_move_cell_indices_d.begin(), 
            OPP_thrust_move_cell_indices_d.begin() + OPP_move_count_h, send_part_cell_idx_hv.begin());
    // opp_profiler->end("Mv_Pack1");

    // opp_profiler->start("Mv_Pack2");
    // enrich the particles to communicate with the correct external cell index and mpi rank
    std::map<int, opp_particle_comm_data>& set_part_com_data = opp_part_comm_neighbour_data[set];
    for (int index = 0; index < OPP_move_count_h; index++)
    {
        // int particle_index = OPP_thrust_move_indices_h[index];
        int map0idx = send_part_cell_idx_hv[index];

        auto it = set_part_com_data.find(map0idx);
        if (it == set_part_com_data.end())
        {
            opp_printf("opp_part_pack_device", 
                "Error: cell %d cannot be found in opp_part_comm_neighbour_data map [%d/%d]", 
                map0idx, index, OPP_move_count_h);
            continue; // unlikely, need opp_abort() instead!
        }

        opp_part_mark_move(set, index, it->second); // it->second is the local cell index in foreign rank
    }
    // opp_profiler->end("Mv_Pack2");

    // opp_profiler->start("Mv_Pack3");
    std::map<int, std::vector<char>> move_dat_data_map;

    // download the particles to send
    {
        for (auto& dat : *(set->particle_dats)) 
        {
            size_t bytes_to_copy = (OPP_move_count_h * dat->size);
            
            auto& move_dat_data = move_dat_data_map[dat->index];
            move_dat_data.resize(bytes_to_copy);

            if (strcmp(dat->type, "double") == 0)
            {
                temp_real_dv.reserve(OPP_move_count_h * dat->dim);
                temp_real_dv.resize(OPP_move_count_h * dat->dim);
                copy_according_to_index(dat->thrust_real, &temp_real_dv, OPP_thrust_move_particle_indices_d, 
                    dat->set->set_capacity, OPP_move_count_h, OPP_move_count_h, dat->dim);
                
                hipMemcpy(move_dat_data.data(), thrust::raw_pointer_cast(&temp_real_dv[0]), 
                    bytes_to_copy, hipMemcpyDeviceToHost);
            }
            else if (strcmp(dat->type, "int") == 0)
            {
                temp_int_dv.reserve(OPP_move_count_h * dat->dim);
                temp_int_dv.resize(OPP_move_count_h * dat->dim);
                copy_according_to_index(dat->thrust_int, &temp_int_dv, OPP_thrust_move_particle_indices_d, 
                    dat->set->set_capacity, OPP_move_count_h, OPP_move_count_h, dat->dim);
                
                hipMemcpy(move_dat_data.data(), thrust::raw_pointer_cast(&temp_int_dv[0]), 
                    bytes_to_copy, hipMemcpyDeviceToHost);
            }
        }      
    }
    // opp_profiler->end("Mv_Pack3");

    // opp_profiler->start("Mv_Pack4");
    opp_part_all_neigh_comm_data* send_buffers = (opp_part_all_neigh_comm_data*)set->mpi_part_buffers;

    // increase the sizes of MPI buffers
    for (auto& move_indices_per_rank : opp_part_move_indices[set->index])
    {
        int send_rank = move_indices_per_rank.first;
        std::vector<opp_part_move_info>& move_indices_vector = move_indices_per_rank.second;

        opp_part_neigh_buffers& send_rank_buffer = send_buffers->buffers[send_rank];
        int64_t required_buffer_size = (move_indices_vector.size() * (int64_t)set->particle_size);

        // resize the export buffer if required
        if (send_rank_buffer.buf_export_index + required_buffer_size >= send_rank_buffer.buf_export_capacity)
        {
            if (send_rank_buffer.buf_export == nullptr)
            {
                send_rank_buffer.buf_export_capacity  = OPP_mpi_part_alloc_mult * required_buffer_size;
                send_rank_buffer.buf_export_index     = 0;
                send_rank_buffer.buf_export           = (char *)opp_host_malloc(send_rank_buffer.buf_export_capacity);

                // opp_printf("opp_part_pack", "alloc buf_export capacity %d", send_rank_buffer.buf_export_capacity);
            }
            else 
            {
                // Assume that there are some particles left already, increase capacity beyond buf_export_index
                send_rank_buffer.buf_export_capacity  = send_rank_buffer.buf_export_index + 
                                                            OPP_mpi_part_alloc_mult * required_buffer_size;
                send_rank_buffer.buf_export           = (char *)opp_host_realloc(send_rank_buffer.buf_export, 
                                                            send_rank_buffer.buf_export_capacity);
                
                // opp_printf("opp_part_pack", "realloc buf_export capacity %d", send_rank_buffer.buf_export_capacity);
            }        
        }
    }
    // opp_profiler->end("Mv_Pack4");

    // opp_profiler->start("Mv_Pack5");
    // iterate over all the ranks and pack to mpi buffers using SOA
    for (auto& move_indices_per_rank : opp_part_move_indices[set->index])
    {
        int send_rank = move_indices_per_rank.first;
        std::vector<opp_part_move_info>& move_indices_vector = move_indices_per_rank.second;
        size_t per_rank_move_count = move_indices_vector.size();

        opp_part_neigh_buffers& send_rank_buffer = send_buffers->buffers[send_rank];

        int64_t displacement = 0;
        for (auto& dat : *(set->particle_dats))
        {
            auto& move_dat_data = move_dat_data_map[dat->index];

            int64_t dat_size = (int64_t)dat->size;
            int64_t element_size = (int64_t)(dat->size / dat->dim);

            if (dat->is_cell_index)
            {
                for (const auto& move_info : move_indices_vector)
                {
                    // we need to copy the cell index of the foreign rank, to correctly unpack in the foreign rank
                    memcpy(&(send_rank_buffer.buf_export[send_rank_buffer.buf_export_index + displacement]), 
                        &move_info.foreign_cell_index, dat->size);
                 
                    displacement += dat_size;
                }
            }
            else
            {
                for (int d = 0; d < dat->dim; d++) 
                {
                    for (const auto& move_info : move_indices_vector)
                    {
                        // copy the multi dimensional dat value to the send buffer
                        memcpy(&(send_rank_buffer.buf_export[send_rank_buffer.buf_export_index + displacement]), 
                            &(move_dat_data[(d * OPP_move_count_h + move_info.local_index) * element_size]), 
                            element_size);
                        
                        displacement += element_size;
                    }
                }                
            }
        }

        send_rank_buffer.buf_export_index = (int64_t)(set->particle_size * move_indices_vector.size()); // Not used
        (send_buffers->export_counts)[send_rank] = (int64_t)move_indices_vector.size();

        move_indices_vector.clear();
    }
    // opp_profiler->end("Mv_Pack5");

    // opp_profiler->start("Mv_Pack6");
    // This particle is already packed, hence needs to be removed from the current rank
    CopyMaxCellIndexFunctor copyMaxCellIndexFunctor((int*)set->mesh_relation_dat->data_d);
    thrust::for_each(OPP_thrust_move_particle_indices_d.begin(), OPP_thrust_move_particle_indices_d.begin() + OPP_move_count_h, 
        copyMaxCellIndexFunctor);
    // opp_profiler->end("Mv_Pack6");

    opp_profiler->end("Mv_Pack");
#endif

    if (OPP_DBG) opp_printf("opp_part_pack_device", "end");
}

//*******************************************************************************
void opp_part_unpack_device(opp_set set)
{
    if (OPP_DBG) opp_printf("opp_part_unpack_device", "set [%s]", set->name);

#ifdef USE_MPI
    opp_profiler->start("Mv_Unpack");

    std::vector<opp_dat>& particle_dats = *(set->particle_dats);
    int64_t num_new_particles = 0;

    opp_part_all_neigh_comm_data* recv_buffers = (opp_part_all_neigh_comm_data*)set->mpi_part_buffers;
    std::vector<int>& neighbours = recv_buffers->neighbours;

    // count the number of particles to be received from all ranks
    for (size_t i = 0; i < neighbours.size(); i++)
    {
        int neighbour_rank = neighbours[i];
        num_new_particles += (recv_buffers->import_counts)[neighbour_rank];
    }

    if (num_new_particles > 0)
    {
        int64_t new_part_index = (int64_t)(set->size);

        opp_increase_particle_count(set, (int)num_new_particles);

        // create a continuous memory in host to copy to device
        std::vector<char*> temp_data_vec;
        for (size_t d = 0; d < particle_dats.size(); d++)
        {
            opp_dat dat = particle_dats[d];
            char *temp_data = (char *)opp_host_malloc(dat->size * num_new_particles * sizeof(char));
            temp_data_vec.push_back(temp_data);
        }

        int current_recv_count = 0;
        for (int i = 0; i < (int)neighbours.size(); i++)
        {
            int recv_rank = neighbours[i];

            opp_part_neigh_buffers& receive_rank_buffer = recv_buffers->buffers[recv_rank];

            int64_t receive_count = recv_buffers->import_counts[recv_rank];
            int64_t displacement = 0;

            for (size_t dat_idx = 0; dat_idx < particle_dats.size(); dat_idx++)
            {
                opp_dat dat = particle_dats[dat_idx];
                char *temp_data = temp_data_vec[dat_idx];

                int element_size = dat->size / dat->dim;

                for (int i = 0; i < dat->dim; i++) 
                {
                    memcpy(&(temp_data[element_size * (i * num_new_particles + current_recv_count)]), 
                        &(receive_rank_buffer.buf_import[displacement]), element_size * receive_count);

                    displacement += element_size * receive_count; 
                }                
            }

            // for (size_t d = 0; d < particle_dats.size(); d++)
            // {
            //     opp_dat dat = particle_dats[d];
            //     char *temp_data = temp_data_vec[d];

            //     int element_size = dat->size / dat->dim;

            //     for (int i = 0; i < dat->dim; i++) 
            //     {
            //         for (int64_t j = 0; j < receive_count; j++) 
            //         {
            //             int64_t tmp_index = element_size * i * num_new_particles + element_size * (j + current_recv_count);
            //             int64_t recv_index = displacement + dat->size * j + element_size * i;

            //             for (int c = 0; c < element_size; c++) 
            //             {
            //                 temp_data[tmp_index + c] = receive_rank_buffer.buf_import[recv_index + c];
            //             }
            //         }
            //     }

            //     displacement += dat->size * receive_count;               
            // }

            // current_recv_count += receive_count;

            current_recv_count += receive_count;
        }

        // copy to device
        for (size_t dat_idx = 0; dat_idx < particle_dats.size(); dat_idx++)
        {
            opp_dat dat = particle_dats[dat_idx];
            char *temp_data = temp_data_vec[dat_idx];

            size_t bytes_to_copy_per_dim = num_new_particles * dat->size / dat->dim;
            int element_size = dat->size / dat->dim;

            for (int64_t d = 0; d < dat->dim; d++) 
            {     
                size_t data_d_offset = (new_part_index + d * set->set_capacity) * element_size;
                size_t data_h_offset = d * num_new_particles * element_size;
    
                cutilSafeCall(hipMemcpy((dat->data_d + data_d_offset), (temp_data + data_h_offset), 
                                    bytes_to_copy_per_dim, hipMemcpyHostToDevice));       
            }
        }

        cutilSafeCall(hipDeviceSynchronize());

        for (auto& char_ptr : temp_data_vec)
            opp_host_free(char_ptr);
        temp_data_vec.clear();
    }

    opp_profiler->end("Mv_Unpack");
#endif

    if (OPP_DBG) opp_printf("opp_part_unpack_device", "END");    
}

//*******************************************************************************


__global__ void copy_intX(const int* in_dat_d, int* out_dat_d, const int* indices,
                    const int in_stride, const int out_stride, const int dim, const int size) 
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) 
    {
        const int idx = indices[tid];
        for (int d = 0; d < dim; d++)
        {
            out_dat_d[tid + d * out_stride] = in_dat_d[idx + d * in_stride];

            // printf("copting index=%d value %d [d=%d] to out_index=%d\n", idx, in_dat_d[idx + d * in_stride], d, tid + d * out_stride);
        }
    }
}

__global__ void copy_doubleX(const double* in_dat_d, double* out_dat_d, const int* indices, 
                    const int in_stride, const int out_stride, const int dim, const int size) 
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) 
    {
        const int idx = indices[tid];
        for (int d = 0; d < dim; d++)
        {
            out_dat_d[tid + d * out_stride] = in_dat_d[idx + d * in_stride];
        }
    }
}

__global__ void copy_doubleY(const double* in_dat_d, double* out_dat_d, int in_stride, 
                                    int out_stride, int dim, int size) 
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) 
    {
        for (int d = 0; d < dim; d++)
        {
            out_dat_d[tid + d * out_stride] = in_dat_d[tid + d * in_stride];
        }
    }
}
__global__ void copy_intY(const int* in_dat_d, int* out_dat_d, int in_stride, 
                                    int out_stride, int dim, int size, int x) 
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) 
    {
        for (int d = 0; d < dim; d++)
        {
            // printf("unpacking %d index=%d value %d [d=%d] to out_index=%d\n", x, tid, in_dat_d[tid + d * in_stride], d, tid + d * out_stride);
            out_dat_d[tid + d * out_stride] = in_dat_d[tid + d * in_stride];
        }
    }
}

__global__ void setArrayToMaxCID(int* array, int* indices, int size) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        int idx = indices[tid];
        array[idx] = MAX_CELL_INDEX;
    }
}

std::map<int, thrust::host_vector<OPP_INT>> particle_indices_hv;    // particle ids to send, arrange according to rank
std::map<int, thrust::host_vector<OPP_INT>> cell_indices_hv;        // cellid in the foreign rank, arrange according to rank
std::map<int, thrust::device_vector<OPP_INT>> particle_indices_dv;
std::map<int, thrust::device_vector<char>> send_data;
std::map<int, thrust::device_vector<char>> recv_data;
const int threads = 64;
const double opp_comm_buff_resize_multiple = 1.5;

// Cannot use multiple packs before sending them, if opp_part_pack() is called multiple times with PACK_SOA, 
// the communication data may get currupted
//*******************************************************************************
void opp_part_pack_and_exchange_cuda_direct(opp_set set)
{
    if (OPP_DBG) opp_printf("opp_part_pack_and_exchange_cuda_direct", "OPP_move_count_h %d", OPP_move_count_h);

#ifdef USE_MPI
    opp_profiler->start("Mv_PackExDir");

    std::map<int, hipStream_t> streams;

    hipStreamCreate(&(streams[-1]));
    hipStreamCreate(&(streams[-2]));

    thrust::host_vector<int> tmp_cell_indices_hv(OPP_move_count_h);
    hipMemcpyAsync(thrust::raw_pointer_cast(tmp_cell_indices_hv.data()), 
            thrust::raw_pointer_cast(OPP_thrust_move_cell_indices_d.data()),
            OPP_move_count_h * sizeof(int), hipMemcpyDeviceToHost, streams[-1]);
    
    thrust::host_vector<int> tmp_particle_indices_hv(OPP_move_count_h);
    hipMemcpyAsync(thrust::raw_pointer_cast(tmp_particle_indices_hv.data()), 
            thrust::raw_pointer_cast(OPP_thrust_move_particle_indices_d.data()),
            OPP_move_count_h * sizeof(int), hipMemcpyDeviceToHost, streams[-2]);

    opp_part_all_neigh_comm_data* mpi_buffers = (opp_part_all_neigh_comm_data*)set->mpi_part_buffers;
    const std::vector<int>& neighbours = mpi_buffers->neighbours;
    const int neighbour_count = neighbours.size();
    mpi_buffers->total_recv = 0;
    for (auto it = mpi_buffers->import_counts.begin(); it != mpi_buffers->import_counts.end(); it++)
        it->second = 0;
    
    for (auto it = particle_indices_hv.begin(); it != particle_indices_hv.end(); it++) it->second.clear();
    for (auto it = cell_indices_hv.begin(); it != cell_indices_hv.end(); it++) it->second.clear();
    for (auto it = particle_indices_dv.begin(); it != particle_indices_dv.end(); it++) it->second.clear();
    for (auto it = send_data.begin(); it != send_data.end(); it++) it->second.clear();
    for (auto it = recv_data.begin(); it != recv_data.end(); it++) it->second.clear();

    mpi_buffers->recv_req.clear();
    mpi_buffers->send_req.clear();
    std::vector<MPI_Request> send_req_count(neighbour_count);
    std::vector<MPI_Request> recv_req_count(neighbour_count);
    double total_send_size = 0.0;

    std::map<int, opp_particle_comm_data>& set_part_com_data = opp_part_comm_neighbour_data[set];

    hipStreamSynchronize(streams[-1]);
    hipStreamSynchronize(streams[-2]);

    // enrich and arrange the particles to communicate with the correct external cell index and mpi rank
    for (int index = 0; index < OPP_move_count_h; index++)
    {
        auto it = set_part_com_data.find(tmp_cell_indices_hv[index]);
        if (it == set_part_com_data.end()) 
        {
            opp_printf("opp_part_pack_and_exchange_cuda_direct", 
                "Error: cell %d cannot be found in opp_part_comm_neighbour_data map", tmp_cell_indices_hv[index]);
            continue; // unlikely, need opp_abort() instead!
        }

        const auto& comm_data = it->second;
        particle_indices_hv[comm_data.cell_residing_rank].push_back(tmp_particle_indices_hv[index]);
        cell_indices_hv[comm_data.cell_residing_rank].push_back(comm_data.local_index); // convert cid to local cid of recv rank
    }
    
    // copy particle_indices_dv to device asynchronously 
    for (const auto& x : particle_indices_hv)
    {
        const int rank = x.first;
        hipStreamCreate(&(streams[rank])); 
        const size_t tmp_cpy_size = x.second.size();

        if (tmp_cpy_size > particle_indices_dv[rank].capacity()) 
            particle_indices_dv[rank].reserve(tmp_cpy_size * opp_comm_buff_resize_multiple);
        particle_indices_dv[rank].resize(tmp_cpy_size);

        hipMemcpyAsync(thrust::raw_pointer_cast(particle_indices_dv[rank].data()), 
            x.second.data(), (tmp_cpy_size * sizeof(int)), hipMemcpyHostToDevice, streams[rank]);
        
        mpi_buffers->export_counts[rank] = tmp_cpy_size;
    }

    // send/receive send_counts to/from all immediate neighbours
    for (int i = 0; i < neighbour_count; i++)
    {
        const int64_t& send_count = mpi_buffers->export_counts[neighbours[i]];
        MPI_Isend((void*)&send_count, 1, MPI_INT64_T, neighbours[i], MPI_COUNT_EXCHANGE, 
            OPP_MPI_WORLD, &(send_req_count[i]));

        const int64_t& recv_count = mpi_buffers->import_counts[neighbours[i]];
        MPI_Irecv((void*)&recv_count, 1, MPI_INT64_T, neighbours[i], MPI_COUNT_EXCHANGE, 
            OPP_MPI_WORLD, &(recv_req_count[i]));
    }

    // pack the send data to device memory arranged according to rank asynchronously
    for (auto& x : particle_indices_dv)
    {
        const int send_rank = x.first;
        const int64_t particle_count = x.second.size();
        const int64_t send_bytes = particle_count * set->particle_size;
        auto& send_data_dv = send_data[send_rank];

        if (send_bytes > send_data_dv.capacity()) 
            send_data_dv.reserve(send_bytes * opp_comm_buff_resize_multiple);
        send_data_dv.resize(send_bytes);

        char* send_buff = (char*)thrust::raw_pointer_cast(send_data_dv.data());
    
        int* particle_indices = (int*)thrust::raw_pointer_cast(particle_indices_dv[send_rank].data());
        const int nblocks = (particle_count - 1) / threads + 1;
        int64_t offset = 0;

        // thrust::host_vector<int> h_vec = particle_indices_dv[send_rank];
        // std::string log = "";
        // for (int i = 0; i < h_vec.size(); ++i) log += std::to_string(h_vec[i]) + " ";
        // opp_printf("TO_SEND", "%s", log.c_str());

        hipStreamSynchronize(streams[send_rank]);

        for (auto& dat : *(set->particle_dats)) 
        {
            const int64_t dat_bytes_to_copy = (particle_count * dat->size);

            if (dat->is_cell_index)
            {
                // cell indices relative to the receiving rank is copied here
                hipMemcpyAsync((send_buff + offset), 
                    (char*)cell_indices_hv[send_rank].data(), dat_bytes_to_copy, 
                    hipMemcpyHostToDevice, streams[send_rank]);
            }
            else if (strcmp(dat->type, "double") == 0)
            {
                copy_doubleX<<<nblocks,threads,0,streams[send_rank]>>> (
                    (OPP_REAL*)dat->data_d,
                    (OPP_REAL*)(send_buff + offset),
                    particle_indices,
                    set->set_capacity, particle_count,
                    dat->dim, particle_count);
            }
            else if (strcmp(dat->type, "int") == 0)
            {
                copy_intX<<<nblocks,threads,0,streams[send_rank]>>> (
                    (OPP_INT*)dat->data_d,
                    (OPP_INT*)(send_buff + offset),
                    particle_indices,
                    set->set_capacity, particle_count,
                    dat->dim, particle_count);
            }
            else
            {
                opp_printf("", "Error: %s type unimplemented in opp_part_pack_and_exchange_cuda_direct", dat->type);
                opp_abort("datatype not implemented in opp_part_pack_and_exchange_cuda_direct");
            }

            offset += dat_bytes_to_copy;
        }
    }

    // since move particles ids are extracted already, mark cell index as MAX_CELL_ID to remove from current rank
    const int nblocks = (OPP_move_count_h - 1) / threads + 1;
    setArrayToMaxCID<<<nblocks,threads>>> (
        (OPP_INT*)set->mesh_relation_dat->data_d,
        (OPP_INT*)thrust::raw_pointer_cast(OPP_thrust_move_particle_indices_d.data()),
        OPP_move_count_h);

    // send the particle data only to immediate neighbours
    for (int i = 0; i < neighbour_count; i++)
    {
        const int send_rank = neighbours[i];
        const int64_t send_count = mpi_buffers->export_counts[send_rank];

        if (send_count <= 0) {
            if (OPP_DBG) opp_printf("opp_part_pack_and_exchange_cuda_direct", "nothing to send to rank %d", send_rank);
            continue;
        }
        else {   
            char* send_buff = (char*)thrust::raw_pointer_cast(send_data[send_rank].data());
            if (OPP_DBG) 
                opp_printf("opp_part_pack_and_exchange_cuda_direct", "sending %lld particle/s (size: %lld) to rank %d | %p", 
                send_count, (int64_t)(send_count*set->particle_size), send_rank, send_buff);
        }

        MPI_Request req;
        const int64_t send_size = set->particle_size * send_count;

        hipStreamSynchronize(streams[send_rank]); // wait till cuda aync copy is done

        char* send_buff = (char*)thrust::raw_pointer_cast(send_data[send_rank].data());
        MPI_Isend(send_buff, send_size, MPI_CHAR, send_rank, MPI_TAG_PART_EX, OPP_MPI_WORLD, &req);
        mpi_buffers->send_req.push_back(req);

        total_send_size += (send_size * 1.0f);
    }

    // wait for the counts to receive only from neighbours
    MPI_Waitall(neighbour_count, &recv_req_count[0], MPI_STATUSES_IGNORE);

    // create/resize data structures and receive particle data from neighbours
    for (int i = 0; i < neighbour_count; i++)
    {
        const int recv_rank = neighbours[i];
        const int64_t recv_bytes = (int64_t)set->particle_size * mpi_buffers->import_counts[recv_rank];
        mpi_buffers->total_recv += mpi_buffers->import_counts[recv_rank];

        if (recv_bytes <= 0)
        {
            if (OPP_DBG) 
                opp_printf("opp_part_pack_and_exchange_cuda_direct", "nothing to receive from rank %d", recv_rank);
            continue;
        }

        auto& recv_data_dv = recv_data[recv_rank];
        if (recv_bytes > recv_data_dv.capacity()) 
            recv_data_dv.reserve(recv_bytes * opp_comm_buff_resize_multiple);
        recv_data_dv.resize(recv_bytes);
        
        MPI_Request req;
        MPI_Irecv((char*)thrust::raw_pointer_cast(recv_data_dv.data()), recv_bytes, MPI_CHAR, 
            recv_rank, MPI_TAG_PART_EX, OPP_MPI_WORLD, &req);
        mpi_buffers->recv_req.push_back(req);
    }

    // reset the export counts for another iteration
    for (auto it = mpi_buffers->export_counts.begin(); it != mpi_buffers->export_counts.end(); it++)
    {
        it->second = 0; // make the export count to zero for the next iteration
        mpi_buffers->buffers[it->first].buf_export_index = 0; // make export indices to zero for next iteration
    }

    // for (const auto& x : streams) hipStreamDestroy(x.second);
    cutilSafeCall(hipDeviceSynchronize());

    opp_profiler->end("Mv_PackExDir");
#endif

    if (OPP_DBG) opp_printf("opp_part_pack_device_direct", "end");
}


void opp_part_unpack_device_direct(opp_set set)
{
    if (OPP_DBG) opp_printf("opp_part_unpack_device_direct", "set [%s] size %d", set->name, set->size);

#ifdef USE_MPI
    opp_profiler->start("Mv_UnpackDir");

    opp_part_all_neigh_comm_data* recv_buffers = (opp_part_all_neigh_comm_data*)set->mpi_part_buffers;
    const auto& neighbours = recv_buffers->neighbours;
    int64_t num_new_particles = 0;
    std::map<int,int64_t> particle_start;

    // count the number of particles to be received from all ranks
    for (size_t i = 0; i < neighbours.size(); i++)
    {
        const int rank = neighbours[i];

        num_new_particles += (recv_buffers->import_counts)[rank];
        if (i == 0) 
            particle_start[i] = set->size;
        else 
            particle_start[i] = particle_start[i-1] + (recv_buffers->import_counts)[neighbours[i-1]];
    }

    if (num_new_particles > 0)
    {
        opp_increase_particle_count(set, (int)num_new_particles);

        for (int i = 0; i < (int)neighbours.size(); i++)
        {
            const int recv_rank = neighbours[i];
            const int recv_count = (int)recv_buffers->import_counts[recv_rank];

            if (recv_count <= 0) continue;

            char* recv_buff = (char*)thrust::raw_pointer_cast(recv_data[recv_rank].data()); 
            const int nblocks = (recv_count - 1) / threads + 1;
            int64_t offset = 0;

            // opp_printf("opp_part_unpack_device_direct", "recv count %d || to dat starting from %lld", 
            //     recv_count, particle_start[i]);

            for (auto& dat : *(set->particle_dats)) 
            {
                const int64_t dat_bytes = (recv_count * dat->size);
                const int64_t dat_per_dim_size = dat->size / dat->dim;

                if (strcmp(dat->type, "double") == 0)
                {
                    copy_doubleY<<<nblocks,threads>>> (
                        (OPP_REAL*)(recv_buff + offset),
                        (OPP_REAL*)(dat->data_d + particle_start[i] * dat_per_dim_size),
                        recv_count, set->set_capacity, dat->dim, recv_count);
                }
                else if (strcmp(dat->type, "int") == 0)
                {
                    int x = 0;

                    if (strcmp(dat->name, "p_index") == 0) x = 111;

                    copy_intY<<<nblocks,threads>>> (
                        (OPP_INT*)(recv_buff + offset),
                        (OPP_INT*)(dat->data_d + particle_start[i] * dat_per_dim_size),
                        recv_count, set->set_capacity, dat->dim, recv_count, x);
                }
                else
                {
                    opp_printf("", "Error: %s type unimplemented in opp_part_unpack_device_direct", dat->type);
                    opp_abort("datatype not implemented in opp_part_unpack_device_direct");
                }

                offset += dat_bytes;
            }         
        }

        cutilSafeCall(hipDeviceSynchronize());
    }

    opp_profiler->end("Mv_UnpackDir");
#endif

    if (OPP_DBG) opp_printf("opp_part_unpack_device_direct", "END");    
}





bool opp_finalize_particle_move(opp_set set)
{ 
    opp_profiler->start("Mv_Finalize");

    cutilSafeCall(hipDeviceSynchronize());

    OPP_move_count_h = 0;
    cutilSafeCall(hipMemcpy(&OPP_move_count_h, OPP_move_count_d, sizeof(int), 
                    hipMemcpyDeviceToHost));

    cutilSafeCall(hipMemcpy(&(set->particle_remove_count), set->particle_remove_count_d, 
                    sizeof(int), hipMemcpyDeviceToHost));

    if (OPP_DBG)
        opp_printf("opp_finalize_particle_move", "set [%s][%d] remove_count [%d] move count [%d]", 
            set->name, set->size, set->particle_remove_count, OPP_move_count_h);

#ifdef USE_MPI
    // At this stage, particles of device is clean
    if (OPP_gpu_direct)
    {
        opp_part_pack_and_exchange_cuda_direct(set);
    }
    else
    {
        // download only the required particles to send and pack them in rank based mpi buffers
        opp_part_pack_device(set);

        // send the counts and send the particle data  
        opp_part_exchange(set); 
    }
#endif

    opp_profiler->start("Mv_fill");
    if (set->particle_remove_count > 0)
    {
        set->size -= set->particle_remove_count;

        if (OPP_fill_type == OPP_HoleFill_All || 
            (OPP_fill_type == OPP_Sort_Periodic || OPP_fill_type == OPP_Shuffle_Periodic) && 
                (OPP_main_loop_iter % OPP_fill_period != 0 || OPP_comm_iteration != 0))
        {
            if (OPP_DBG) 
                opp_printf("opp_finalize_particle_move", "hole fill set [%s]", set->name);
            
            particle_hole_fill_device(set);
        }
        else if (OPP_fill_type == OPP_Sort_All || OPP_fill_type == OPP_Sort_Periodic)
        {
            if (OPP_DBG)
                opp_printf("opp_finalize_particle_move", "sort set [%s]", set->name);
            
            opp_particle_sort(set);
        }
        else if (OPP_fill_type == OPP_Shuffle_All || OPP_fill_type == OPP_Shuffle_Periodic)
        {
            if (OPP_DBG) 
                opp_printf("opp_finalize_particle_move", "shuffle set [%s]", set->name);
            
            particle_sort_device(set, true); // true will shuffle the particles
        }
    }
    opp_profiler->end("Mv_fill");

#ifdef USE_MPI
    if (opp_part_check_all_done(set))
    {
        if (OPP_max_comm_iteration < OPP_comm_iteration)
            OPP_max_comm_iteration = OPP_comm_iteration;

        OPP_comm_iteration = 0; // reset for the next par loop
        
        cutilSafeCall(hipDeviceSynchronize());
        opp_profiler->end("Mv_Finalize");
        return false; // all mpi ranks do not have anything to communicate to any rank
    }

    opp_part_wait_all(set); // wait till all the particles are communicated

    if (OPP_DBG)
        opp_printf("opp_finalize_particle_move", "set [%s] size prior unpack %d", set->name, set->size);
    
    cutilSafeCall(hipDeviceSynchronize());

    // increase the particle count if required and unpack the communicated particles to separate dats
    if (OPP_gpu_direct)
    {
        opp_part_unpack_device_direct(set);  
    }
    else
    {
        opp_part_unpack_device(set);    
    }

    OPP_iter_start = set->size - set->diff;
    OPP_iter_end   = set->size;  

    OPP_comm_iteration++;  

    opp_profiler->end("Mv_Finalize");

    return true;
#else
    return false;
#endif
}